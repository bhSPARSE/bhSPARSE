#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
// < A CUDA/OpenCL General Sparse Matrix-Matrix Multiplication Program >
//
// < See paper:
// Weifeng Liu and Brian Vinter, "An Efficient GPU General Sparse
// Matrix-Matrix Multiplication for Irregular Data," Parallel and
// Distributed Processing Symposium, 2014 IEEE 28th International
// (IPDPS '14), pp.370-381, 19-23 May 2014
// for details. >
//////////////////////////////////////////////////////////////////////////

#include "ref.h"

#include <cusp/multiply.h>
#include <cusp/detail/host/reference/csr.h>

ref::ref()
{
}

template<class I, class T>
void ref::csr_sort_indices(const I n_row,
                      const I Ap[],
                            I Aj[],
                            T Ax[])
{
    std::vector< std::pair<I,T> > temp;

    for(I i = 0; i < n_row; i++){
        I row_start = Ap[i];
        I row_end   = Ap[i+1];

        temp.clear();

        for(I jj = row_start; jj < row_end; jj++){
            temp.push_back(std::make_pair(Aj[jj],Ax[jj]));
        }

        std::sort(temp.begin(),temp.end(),kv_pair_less<I,T>);

        for(I jj = row_start, n = 0; jj < row_end; jj++, n++){
            Aj[jj] = temp[n].first;
            Ax[jj] = temp[n].second;
        }
    }
}


void ref::compData(CSRHost A, CSRHost B, int m, int nnzC, index_type *csrRowPtrC, index_type *csrColIndC, value_type *csrValC)
{
    cout << endl << "Checking correctness ..." << endl;

    COODevice dAcoo = A;
    COODevice dBcoo = B;
    COODevice dCcoo;

    cusp::multiply(dAcoo, dBcoo, dCcoo);

    CSRHost C = dCcoo;

    // check it on CPU, but CUSP v0.4.0's CPU SpGEMM seems incorrect
    //    cusp::coo_matrix<index_type, value_type, cusp::host_memory> Ccoo;
    //    Ccoo = C;
    //    Ccoo.sort_by_row_and_column();
    //    C = Ccoo;

    csr_sort_indices<index_type, value_type>(m, &C.row_offsets[0], &C.column_indices[0], &C.values[0]);

    // check nnzC
    if (C.num_entries == nnzC)
        cout << "nnzC = " << nnzC << ". PASS!" << endl;
    else
    {
        cout << "nnzC = " << nnzC << ", CUSP's nnzC = " << C.num_entries << ". NO PASS!" << endl;
        return;
    }

    // check csrRowPtrC
    int err_count = 0;
    for (int i = 0; i <= m; i++)
    {
        if (C.row_offsets[i] != csrRowPtrC[i])
            err_count++;
    }
    if (!err_count)
        cout << "RowPtrC PASS!" << endl;
    else
    {
        cout << "RowPtrC NO PASS!" << endl;
        return;
    }

    // check csrColIndC and csrValC
    err_count = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = csrRowPtrC[i]; j < csrRowPtrC[i+1]; j++)
        {
            if (C.column_indices[j] != csrColIndC[j] ||
                    fabs((double)C.values[j] - (double)csrValC[j]) > fabs(0.1 * (double)C.values[j]) )
            {
                err_count++;
//                cout << "Row = " << i
//                     << " CUSP: ColIndC = " << C.column_indices[j]
//                     << " ValC = " << C.values[j]
//                     << " BHSPARSE: ColIndC = " << csrColIndC[j]
//                     << " ValC = " << csrValC[j]
//                     << endl;
            }
        }
    }

    if (!err_count)
        cout << "ColIndC/csrValC PASS!" << endl;
    else
        cout << "ColIndC/csrValC NO PASS! #err = " << err_count << endl;
}
